#include "hip/hip_runtime.h"
#include "cuda_v1_shim.h"
#include <stdio.h>
#include <stdint.h>
#include <mutex>
#include <thread>
#include <vector>
#include <string>
#include <chrono>
#include <condition_variable>
#include <cstring>
#include <unordered_map>
#include <fstream>
#include <sstream>
#include <unistd.h>
#include <iomanip>
#include <unordered_set>
#include <execinfo.h>
#include <signal.h>

static inline void _drv_check(hipError_t r, const char* call,
                              const char* file, int line) {
    if (r != hipSuccess) {
        const char *name = "unknown";
        hipDrvGetErrorName(r, &name);
        fprintf(stderr, "[DRV] %s:%d %s -> %s\n", file, line, call, name);
    }
}
#define DRV(call)  _drv_check((call), #call, __FILE__, __LINE__)

// Simple SIGSEGV handler to see where we crash
static void segv_handler(int) {
    void* bt[32];
    int n = backtrace(bt, 32);
    backtrace_symbols_fd(bt, n, STDERR_FILENO);
    _exit(1);
}





#include "nvbit_tool.h"
#include "nvbit.h"

#ifdef DEBUG
#define DBG(msg, ...)                                                      \
    printf("[DBG] %s:%d " msg "\n", __FUNCTION__, __LINE__, ##__VA_ARGS__); \
    fflush(stdout);
#else
#define DBG(msg, ...)
#endif

struct KernelArgLog {
    hipCtx_t                    ctx;
    std::string                  kernel_name;
    std::vector<hipDeviceptr_t>     dev_ptrs;
    std::vector<size_t>          sizes;
    std::string                  sass_file;
    int                          grid_x;
    int                          block_x;
};

static std::mutex                log_mtx;
static std::condition_variable   cv;
static std::vector<KernelArgLog> pending;
static bool                      keep_running = true;
static std::unordered_map<hipDeviceptr_t, size_t> alloc_size;
static std::unordered_map<hipDeviceptr_t, size_t> memcpy_size;
static std::unordered_set<hipDeviceptr_t> freed_ptrs;

static std::string dump_sass_to_tmp(const std::vector<Instr*>& instrs, const std::string& kernel) {
    std::ostringstream fname;
    fname << "/home/vic/Dev/sass_ptx_parser/tmp/" << kernel << "-" << getpid() << ".sass";
    {std::ofstream out(fname.str(), std::ios::out | std::ios::trunc);
    for (Instr* i : instrs) {
        out << i->getSass() << '\n';
    }}
    return fname.str();
}

static void launch_analyser(const std::string& sass_path,
                            int grid, int block, hipDeviceptr_t base)
{
    std::ostringstream cmd;
    cmd << "python3 /home/vic/Dev/sass_ptx_parser/ptx_parser/main.py "
        << std::quoted(sass_path) << ' '
        << "--grid "  << grid << ' '
        << "--block " << block << ' '
        << "--base 0x" << std::hex << base << ' '
        << "--json_out " << std::quoted(sass_path) << ".json"
        << " &";                    // run detached
    
    std::cout << "[DBG] Dumping sass to" << sass_path;
    std::system(cmd.str().c_str());
}

static void mem_dumper() {
    DBG("[DBG] mem_dumper thread started");
    while (true) {
        KernelArgLog job;
        
        {
            std::unique_lock<std::mutex> lk(log_mtx);
            cv.wait(lk, []{ return !pending.empty() || !keep_running; });
            if (!keep_running && pending.empty()) {
                DBG("[DBG] mem_dumper thread exiting");
                return;  
            }  

            DBG("Taking job from queue, queue size=%zu", pending.size());
            job = pending.back();  // Use copy instead of move
            pending.pop_back();
        }

        DBG("Processing job for kernel '%s', pointers=%zu, sass_file='%s'", 
            job.kernel_name.c_str(), job.dev_ptrs.size(), job.sass_file.c_str());
        
        if (hipCtxSetCurrent(job.ctx) != hipSuccess) {
            DBG("Context 0x%p no longer valid", job.ctx);
            continue;
        }

        
        for (size_t i = 0; i < job.dev_ptrs.size(); ++i) {
            if (freed_ptrs.count(job.dev_ptrs[i])) {
                DBG("arg[%zu] 0x%llx already freed - skip", i,
                    (unsigned long long)job.dev_ptrs[i]);
                continue;
            }
            hipDeviceptr_t base_live;
            size_t bytes_live;
            //if (DRV(hipMemGetAddressRange(&base_live,&bytes_live, job.dev_ptrs[i])), false) {
            //    DBG("hipMemGetAddressRange failed -> skip");
            //    continue;
            //}
            if (hipMemGetAddressRange(&base_live, &bytes_live, job.dev_ptrs[i]) != hipSuccess) {
                DBG("arg[%zu] 0x%llx no longer valid - skip", i,
                    (unsigned long long)job.dev_ptrs[i]);
                continue;
            }
            size_t want = std::min(job.sizes[i], bytes_live);
            if (!want) { DBG("arg[%zu] size unknown - skip", i); continue; }

            DBG("copy %zu B from 0x%llx", want,
                (unsigned long long)job.dev_ptrs[i]);

            unsigned mem_type = 0;
            if (hipPointerGetAttribute(&mem_type,
                   HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, job.dev_ptrs[i]) != hipSuccess
                || mem_type != hipMemoryTypeDevice)
                continue;                     

            std::vector<uint8_t> h(want);
            //DRV(hipMemcpyDtoH(h.data(), job.dev_ptrs[i], want));
            if (hipMemcpyDtoH(h.data(), job.dev_ptrs[i], h.size()) == hipSuccess) {
                printf("[MEMDUMP] %s arg[%zu] 0x%llx :", job.kernel_name.c_str(),
                       i, (unsigned long long)job.dev_ptrs[i]);
                for (size_t b = 0; b < h.size() && b < 16; ++b) printf(" %02x", h[b]);
                puts("");
            }
        }
        if (!job.sass_file.empty()) {
            hipDeviceptr_t base = job.dev_ptrs.empty() ? 0 : job.dev_ptrs.front();
            launch_analyser(job.sass_file, job.grid_x,
                             job.block_x, base);
            DBG("sass dumped to %s", job.sass_file.c_str());
        }


    }
}

static std::thread dumper_thread;

extern "C" void nvbit_at_init() {
    //signal(SIGSEGV, segv_handler);
    puts("[NVBIT] arg-logger initialised");
    dumper_thread = std::thread(mem_dumper);
}

extern "C" void nvbit_at_term() {
    {   std::lock_guard<std::mutex> lk(log_mtx);
        keep_running = false;
    }
    cv.notify_all();             
    dumper_thread.join();       
    DBG("[DBG] dumper thread joined");
    puts("[NVBIT] arg-logger exiting");
}

extern "C" void nvbit_at_cuda_event(hipCtx_t ctx,
                                    int is_exit,
                                    nvbit_api_cuda_t cbid,
                                    const char*,
                                    void* params,
                                    hipError_t*) {
    
    //if (!is_exit && (cbid == API_CUDA_cuMemAlloc_v2 || cbid == API_CUDA_cuMemAllocManaged))
    //{
    //    auto *pa = (cuMemAlloc_v2_params*)params;
    //    hipDeviceptr_t gpu = *pa->dptr;
    //    alloc_size[gpu] = pa->bytesize;
    //    alloc_size[*pa->dptr] = pa->bytesize;
    //    DBG("alloc 0x%llx %zu B", (unsigned long long)gpu, pa->bytesize);
    //    return;
    //}

    if (is_exit &&                       // <-- change   !is_exit  → is_exit
        (cbid == API_CUDA_cuMemAlloc_v2 || cbid == API_CUDA_cuMemAllocManaged)) {
        auto *pa = (cuMemAlloc_v2_params*)params;
        hipDeviceptr_t gpu = *pa->dptr;
        if (gpu) alloc_size[gpu] = pa->bytesize;
        return;
    }
    if (!is_exit && (cbid == API_CUDA_cuMemcpyHtoD_v2 || cbid == API_CUDA_cuMemcpyHtoDAsync_v2)) {
        auto *pm = (cuMemcpyHtoD_v2_params*)params;
        memcpy_size[pm->dstDevice] = pm->ByteCount;
        DBG("memcpy H→D 0x%llx %zu B", (unsigned long long)pm->dstDevice, pm->ByteCount);
        return;
    }

    if (!is_exit && cbid == API_CUDA_cuMemFree_v2) {
        auto *pf = (cuMemFree_v2_params*)params;
        freed_ptrs.insert(pf->dptr);
        return;
    }

    if (is_exit) return;
    if (cbid != API_CUDA_cuLaunchKernel &&
        cbid != API_CUDA_cuLaunchKernel_ptsz) return;
    
    DBG("[DBG] hipModuleLaunchKernel intercepted");
    auto* p = (cuLaunchKernel_params*)params;

    KernelArgLog job;
    job.ctx         = ctx;
    job.kernel_name = nvbit_get_func_name(ctx, p->f);

    // get sass from kenrel
    const std::vector<Instr*>& instrs = nvbit_get_instrs(ctx, p->f);
    for (Instr* i : instrs) {
        std::cout << "[" << i->getIdx() << "] "
                  << "Offset: " << i->getOffset() << "\t"
                  << i->getSass() << "\n";
    
    }
    std::string sass_file = dump_sass_to_tmp(instrs, job.kernel_name);

    

    if (hipCtxSetCurrent(ctx) != hipSuccess) {
        fprintf(stderr, "[DRV] hipCtxSetCurrent failed in callback\n");
        return;
    }
    void** kparams = (void**)p->kernelParams;
    if (kparams)
        for (int i = 0; i < 64 && kparams[i]; ++i) {
            uintptr_t host_ptr = (uintptr_t)kparams[i];

            if (host_ptr < 0x100000000000ULL)
            {
                DBG("[DBG] args[%d] looks like scalar - skipping", i);
                //break;
                continue;
            }

            hipDeviceptr_t dev_ptr = 0;
            memcpy(&dev_ptr, kparams[i], sizeof(hipDeviceptr_t));

            if (dev_ptr < 0x700000000000ULL || dev_ptr > 0x7fffffffffffULL) {
                DBG("  arg[%d] 0x%llx outside GPU range - skip",
                    i, (unsigned long long)dev_ptr);
                //break;
                continue;
            }

            hipCtx_t dummy;
            if (hipPointerGetAttribute(&dummy,
                    HIP_POINTER_ATTRIBUTE_CONTEXT, dev_ptr) != hipSuccess) {
                DBG("arg[%d] unowned pointer - stop scan", i);
                //break;
                continue;
            }

            unsigned mem_type = 0;

            if (hipPointerGetAttribute(&mem_type,
                   HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, dev_ptr) != hipSuccess
                || mem_type != hipMemoryTypeDevice) {
                    DBG("[DBG] args[%d] looks like host pointer - skipping", i);
                    //break;
                    continue;                      
                }
           
                size_t sz = 0;
                auto m = memcpy_size.find(dev_ptr);
                if (m != memcpy_size.end()) sz = m->second;
                else {
                    auto a = alloc_size.find(dev_ptr);
                    if (a != alloc_size.end()) sz = a->second;
                }

                if (sz == 0) {
                    hipDeviceptr_t base_ptr = 0;

                    if (hipMemGetAddressRange(&base_ptr, &sz, dev_ptr) != hipSuccess) {
                        DBG("arg[%d] 0x%llx size unknown - skip", i,
                      (unsigned long long)dev_ptr);
                        continue;
                    }
                }

                DBG("arg[%d] dev=0x%llx size=%zu queued", i,
                    (unsigned long long)dev_ptr, sz);

            job.dev_ptrs.push_back(dev_ptr);
            job.sizes.push_back(sz);       
        }
    //hipDeviceptr_t base = job.dev_ptrs.empty() ? 0 : job.dev_ptrs[0];
    //launch_analyser(sass_file, p->gridDimX,
    //                 p->blockDimX, base);
    //DBG("sass dumped to %s", sass_file.c_str());
    job.sass_file = sass_file;
    job.grid_x    = p->gridDimX;
    job.block_x   = p->blockDimX;


    {
        std::lock_guard<std::mutex> lk(log_mtx);
        pending.push_back(job);
    }
    cv.notify_one();               
    DBG("queued");
}