#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void simple_cond_kernel_signed(int* ptr, int condition) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only thread 0 operates
    if (idx == 0) {
        if (condition > 0) {  // Check if condition is positive
            *ptr = 42;        // Write if positive
        } else {
            int dummy = *ptr; // Read if zero or negative (blind read)
        }
    }
}

int main() {
    int *h_data, *d_data;
    int initial_value = 100;
    int condition = 5;  // Change to 0 or negative to test read path
    
    // Allocate host memory
    h_data = (int*)malloc(sizeof(int));
    *h_data = initial_value;
    
    // Allocate device memory
    hipMalloc(&d_data, sizeof(int));
    hipMemcpy(d_data, h_data, sizeof(int), hipMemcpyHostToDevice);
    
    printf("Initial value: %d\n", *h_data);
    printf("Condition: %d (%s)\n", condition, 
           condition > 0 ? "positive, will write 42" : "zero/negative, will read");
    
    // Launch kernel with 1 block, 1 thread
    simple_cond_kernel_signed<<<1, 1>>>(d_data, condition);
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Final value: %d\n", *h_data);
    
    // Cleanup
    hipFree(d_data);
    free(h_data);
    
    return 0;
}