#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void int_one_hot(float* output, short input_val) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < 65536) 
    {
        output[idx] = (idx == (int)input_val) ? 1.0f : 0.0f;
    }
}

int main() {
    short input_value = 1234;
    
    const int output_size = 65536; 
    float* h_output = (float*)malloc(output_size * sizeof(float));
    
    float* d_output;
    hipMalloc(&d_output, output_size * sizeof(float));
    
    hipMemset(d_output, 0, output_size * sizeof(float));
    
    int threads_per_block = 256;
    int blocks = (output_size + threads_per_block - 1) / threads_per_block;
    
    printf("Input value: %d\n", input_value);
    printf("Launching kernel with %d blocks, %d threads per block\n", blocks, threads_per_block);
    
    int_one_hot<<<blocks, threads_per_block>>>(d_output, input_value);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    hipDeviceSynchronize();
    
    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("One-hot encoding verification:\n");
    for (int i = input_value - 5; i <= input_value + 5; i++) 
    {
        if (i >= 0 && i < output_size) 
        {
            printf("output[%d] = %.1f\n", i, h_output[i]);
        }
    }
    
    int count_ones = 0;
    int one_index = -1;
    for (int i = 0; i < output_size; i++) 
    {
        if (h_output[i] == 1.0f) 
        {
            count_ones++;
            one_index = i;
        }
    }
    
    printf("Number of 1.0 values: %d\n", count_ones);
    if (count_ones == 1 && one_index == input_value) 
    {
        printf("SUCCESS: One-hot encoding is correct! Value 1.0 found at index %d\n", one_index);
    }
    else 
    {
        printf("ERROR: One-hot encoding failed!\n");
    }
    
    free(h_output);
    hipFree(d_output);
    
    return 0;
}