#include <hip/hip_runtime.h>
#include <iostream>

__global__ void write(float *out) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    out[global_idx] = 1.0f;
}

int main() {
    const int threads_per_block = 128;
    const int num_blocks = 4;
    const int total_threads = threads_per_block * num_blocks;
    const int size = total_threads * sizeof(float);

    float *d_out;
    hipMalloc(&d_out, size);

    write<<<num_blocks, threads_per_block>>>(d_out);

    float *h_out = new float[total_threads];
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    std::cout << "First 10 values after write kernel:\n";
    for (int i = 0; i < 10; ++i) {
        std::cout << "h_out[" << i << "] = " << h_out[i] << "\n";
    }

    delete[] h_out;
    hipFree(d_out);

    return 0;
}