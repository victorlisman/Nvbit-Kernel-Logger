#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024
#define BLOCK_SIZE 256

__global__ void copy_element_kernel(int* X, int* Y, int i) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only one thread performs the copy operation
    if (idx == 0 && i < N) {
        Y[i] = X[i];
    }
}

int main() {
    int *h_X, *h_Y;  // Host arrays
    int *d_X, *d_Y;  // Device arrays
    int i = 42;      // Index to copy (example)
    
    // Allocate host memory
    h_X = (int*)malloc(N * sizeof(int));
    h_Y = (int*)malloc(N * sizeof(int));
    
    // Initialize arrays
    for (int j = 0; j < N; j++) {
        h_X[j] = j * 2;  // Example values
        h_Y[j] = 0;      // Initialize Y to zeros
    }
    
    // Allocate device memory
    hipMalloc(&d_X, N * sizeof(int));
    hipMalloc(&d_Y, N * sizeof(int));
    
    // Copy data to device
    hipMemcpy(d_X, h_X, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, N * sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE);
    
    printf("Copying element at index %d from X to Y\n", i);
    printf("Before: X[%d] = %d, Y[%d] = %d\n", i, h_X[i], i, h_Y[i]);
    
    copy_element_kernel<<<grid, block>>>(d_X, d_Y, i);
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Copy result back to host
    hipMemcpy(h_Y, d_Y, N * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("After:  X[%d] = %d, Y[%d] = %d\n", i, h_X[i], i, h_Y[i]);
    
    // Verify the copy
    if (h_Y[i] == h_X[i]) {
        printf("SUCCESS: Element copied correctly!\n");
    } else {
        printf("ERROR: Element not copied correctly!\n");
    }
    
    // Cleanup
    free(h_X);
    free(h_Y);
    hipFree(d_X);
    hipFree(d_Y);
    
    return 0;
}