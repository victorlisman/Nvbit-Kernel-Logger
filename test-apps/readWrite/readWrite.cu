#include <hip/hip_runtime.h>
#include <iostream>

// Kernel: each thread copies one float from x to y
__global__ void readWriteKernel(float* x, float* y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    y[idx] = x[idx];
}

int main() {
    int N = 1024;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float* h_x = new float[N];
    float* h_y = new float[N];

    // Initialize input data
    for (int i = 0; i < N; ++i) {
        h_x[i] = static_cast<float>(i);
    }

    // Allocate device memory
    float* d_x;
    float* d_y;
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    // Copy data from host to device
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    // Kernel launch configuration
    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);  // Ceiling division

    // Launch the kernel
    readWriteKernel<<<gridDim, blockDim>>>(d_x, d_y);

    // Copy result back to host
    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    // Validate result
    bool correct = true;
    for (int i = 0; i < N; ++i) {
        if (h_y[i] != h_x[i]) {
            std::cout << "Mismatch at index " << i << ": " << h_y[i] << " != " << h_x[i] << std::endl;
            correct = false;
            break;
        }
    }
    if (correct) {
        std::cout << "Success: all values match!" << std::endl;
    }

    // Free memory
    hipFree(d_x);
    hipFree(d_y);
    delete[] h_x;
    delete[] h_y;

    return 0;
}