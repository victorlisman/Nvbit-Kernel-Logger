#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void simple_loop_kernel_dd(int* iterations_ptr, int* ptr) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
     
    if (idx == 0) {
        int iterations = *iterations_ptr;   
        int i = 0;
        while (i < iterations) {
            *ptr = *ptr + 1;   
            i++;
        }
    }
}

int main() {
    int *h_data, *d_data;
    int *h_iterations, *d_iterations;
    int initial_value = 100;
    int iterations_value = 5;   
    
     
    h_data = (int*)malloc(sizeof(int));
    h_iterations = (int*)malloc(sizeof(int));
    *h_data = initial_value;
    *h_iterations = iterations_value;
    
     
    hipMalloc(&d_data, sizeof(int));
    hipMalloc(&d_iterations, sizeof(int));
    hipMemcpy(d_data, h_data, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_iterations, h_iterations, sizeof(int), hipMemcpyHostToDevice);
    
    printf("Initial value: %d\n", *h_data);
    printf("Loop iterations: %d\n", *h_iterations);
    
     
    simple_loop_kernel_dd<<<1, 1>>>(d_iterations, d_data);
    hipDeviceSynchronize();
    
     
    hipMemcpy(h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Final value: %d\n", *h_data);
    
     
    hipFree(d_data);
    hipFree(d_iterations);
    free(h_data);
    free(h_iterations);
    
    return 0;
}